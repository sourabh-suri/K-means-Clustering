#include "hip/hip_runtime.h"
#include"headerfiles.h"
#include"cudafiles.h"
// Number of threads
#define BLOCK_SIZE 16
#define GRID_SIZE 256
// Class object from opencv: 3D Input Image
Mat input_image;

// num_of_centroids and size on gpu
__constant__ int gpu_num_of_centroids;
__constant__ int gpu_size;

//R,G,B Centroid's triple on gpu
__constant__ int gpu_red_centroid[20];
__constant__ int gpu_green_centroid[20];
__constant__ int gpu_blue_centroid[20];


int size_for_centroids = 0; // num_of_centroids * sizeof(int)
int size_image = 0;  // width * height * sizeof(int)


__global__ void sum_pixels_in_cluster(int *gpu_red,int *gpu_green,int *gpu_blue,int *gpu_sum_red,int *gpu_sum_green,int *gpu_sum_blue,int *gpu_cluster_array,int * gpu_points_in_cluster)
 {
 	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;


 	if(threadID < gpu_size) {
 		int selected_cluster_array = gpu_cluster_array[threadID];
 		int selected_red_val = gpu_red[threadID];
 		int selected_green_val = gpu_green[threadID];
 		int selected_blue_val = gpu_blue[threadID];
 		atomicAdd(&gpu_sum_red[selected_cluster_array], selected_red_val);
 		atomicAdd(&gpu_sum_green[selected_cluster_array], selected_green_val);
 		atomicAdd(&gpu_sum_blue[selected_cluster_array], selected_blue_val);
 		atomicAdd(& gpu_points_in_cluster[selected_cluster_array], 1);
 	}

__global__ void update_centroids(int *gpu_update_red_centroid, int *gpu_update_green_centroid, int *gpu_update_blue_centroid,int* gpu_sum_red, int *gpu_sum_green,int *gpu_sum_blue, int*  gpu_points_in_cluster,int *gpu_flag)
{

 	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
 	if(threadID < gpu_num_of_centroids)
  {
 		int points_in_selected_cluster =  gpu_points_in_cluster[threadID];
 		int sum_red = gpu_sum_red[threadID];
 		int sum_green = gpu_sum_green[threadID];
 		int sum_blue = gpu_sum_blue[threadID];

	gpu_update_red_centroid[threadID] = (int)(sum_red/points_in_selected_cluster);

 		gpu_update_green_centroid[threadID] = (int)(sum_green/points_in_selected_cluster);
 		gpu_update_blue_centroid[threadID] = (int)(sum_blue/points_in_selected_cluster);

 		if(gpu_update_green_centroid[threadID]!=gpu_green_centroid[threadID] || gpu_update_red_centroid[threadID]!=gpu_red_centroid[threadID] || gpu_update_blue_centroid[threadID]!=gpu_blue_centroid[threadID])
 		*gpu_flag=1;
 	}
}
__global__ void pop_gpu_arrays(int *gpu_sum_red,int *gpu_sum_green,int *gpu_sum_blue, int*  gpu_points_in_cluster, int* gpu_update_red_centroid, int* gpu_update_green_centroid, int* gpu_update_blue_centroid )
{

 	int threadID = threadIdx.x + threadIdx.y * blockDim.x;
 	if(threadID < gpu_num_of_centroids)
  {
 		// num_of_centroids long
 		gpu_sum_red[threadID] = 0;
 		gpu_sum_green[threadID] = 0;
 		gpu_sum_blue[threadID] = 0;
 		gpu_update_red_centroid[threadID] = 0;
 		gpu_update_green_centroid[threadID] = 0;
 		gpu_update_blue_centroid[threadID] = 0;
    gpu_points_in_cluster[threadID] = 0;
 	}
}
__global__ void pop_gpu_cluster_array(int *gpu_cluster_array)
{

	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

	if(threadID < gpu_size) {
		gpu_cluster_array[threadID] = 0;
	}
}



__global__ void get_cluster_points(int *gpu_red,int *gpu_green,int *gpu_blue,int *gpu_cluster_array)
{

 	int threadID = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

 	//default min value of distance
 	float min = 1000.0, value;
 	int index = 0;
 	if(threadID < gpu_size) {

 		for(int i = 0; i < gpu_num_of_centroids; i++) {

 			value = sqrtf(powf((gpu_red[threadID]-gpu_red_centroid[i]),2.0) + powf((gpu_green[threadID]-gpu_green_centroid[i]),2.0) + powf((gpu_blue[threadID]-gpu_blue_centroid[i]),2.0));
 			if(value < min){
 				// saving new nearest centroid
 				min = value;
 				// Updating his index
 				index = i;
 			}
 		}
 		gpu_cluster_array[threadID] = index;
 	}
}





int main(int argc, char *argv[])
{
		hipSetDevice(0);
		int *r, *g, *b, *cpu_red_centroid, *cpu_green_centroid, *cpu_blue_centroid;
		int *gpu_red, *gpu_green, *gpu_blue, *gpu_update_red_centroid, *gpu_update_green_centroid, *gpu_update_blue_centroid;
		int *cpu_cluster_Array, *gpu_cluster_array;
    int *gpu_flag, cpu_flag;
		int *sum_red, *sum_green, *sum_blue;
		int *gpu_sum_red, *gpu_sum_green, *gpu_sum_blue;
    int width, height, num_of_centroids, num_of_iteration,size;
    int *cpu_points_in_cluster, *gpu_points_in_cluster;

  // Reading the command line arguments
  printf("[1]loading image...\n");
  input_image = imread(argv[1], IMREAD_COLOR);
  if(! input_image.data )
  {
  cout <<  "Could not open or find the image" << std::endl ;
  return -1;
  }
	num_of_centroids = atoi(argv[2]);
	num_of_iteration = atoi(argv[3]);

	// Allocating memory on CPU
  width = input_image.cols;
  height = input_image.rows;
	size_image = width * height * sizeof(int);
	size_for_centroids = num_of_centroids * sizeof(int);
	size = width * height;
  // 1D pointer arrays
	r = (int*)(malloc(size_image));
	g = (int*)(malloc(size_image));
	b = (int*)(malloc(size_image));
	cpu_red_centroid = (int*)(malloc(size_for_centroids));
	cpu_green_centroid = (int*)(malloc(size_for_centroids));
	cpu_blue_centroid = (int*)(malloc(size_for_centroids));
	cpu_cluster_Array = (int*)(malloc(size_image)); //stores the cluster number for each pixel
	sum_red = (int*)(malloc(size_for_centroids));
	sum_green = (int*)(malloc(size_for_centroids));
	sum_blue = (int*)(malloc(size_for_centroids));
	cpu_points_in_cluster = (int*)(malloc(size_for_centroids));


  // 3D Image spread to three 1D pointer arrays
  for(int i=0;i<input_image.rows;i++)
   {
     for(int j=0;j<input_image.cols;j++)
     {
       Vec3b intensity = input_image.at<Vec3b>(i,j);
       *(b + i*input_image.cols + j) = (int)intensity.val[0];
       *(g + i*input_image.cols + j) = (int)intensity.val[1];
       *(r + i*input_image.cols + j) = (int)intensity.val[2];
     }
  }

	// Setting initial centroids
	initialise_centroids(num_of_centroids, cpu_red_centroid, cpu_green_centroid, cpu_blue_centroid,r,g,b,size);

  // Allocating memory on GPU
  hipMalloc((void**) &gpu_sum_red, size_for_centroids);
  hipMalloc((void**) &gpu_sum_green, size_for_centroids);
  hipMalloc((void**) &gpu_sum_blue, size_for_centroids);
  hipMalloc((void**) &gpu_update_red_centroid, size_for_centroids);
  hipMalloc((void**) &gpu_update_green_centroid, size_for_centroids);
  hipMalloc((void**) &gpu_update_blue_centroid, size_for_centroids);
  hipMalloc((void**) &gpu_cluster_array, size_image);
  hipMalloc((void**) &gpu_red, size_image);
  hipMalloc((void**) &gpu_green, size_image);
  hipMalloc((void**) &gpu_blue, size_image);
  hipMalloc((void**) & gpu_points_in_cluster, size_for_centroids);
  hipMalloc((void**) &gpu_flag, sizeof(int));

  	// copy CPU memory to GPU
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_red_centroid), cpu_red_centroid, size_for_centroids);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_green_centroid), cpu_green_centroid, size_for_centroids);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_blue_centroid), cpu_blue_centroid, size_for_centroids);
  hipMemcpy(gpu_update_red_centroid, cpu_red_centroid,size_for_centroids,hipMemcpyHostToDevice);
  hipMemcpy(gpu_update_green_centroid, cpu_green_centroid,size_for_centroids,hipMemcpyHostToDevice);
  hipMemcpy(gpu_update_blue_centroid, cpu_blue_centroid,size_for_centroids,hipMemcpyHostToDevice);
  hipMemcpy(gpu_cluster_array, cpu_cluster_Array, size_image, hipMemcpyHostToDevice);
  hipMemcpy(gpu_flag,&cpu_flag,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy( gpu_points_in_cluster, cpu_points_in_cluster, size_for_centroids, hipMemcpyHostToDevice);
  hipMemcpy(gpu_red, r, size_image, hipMemcpyHostToDevice);
  hipMemcpy(gpu_green, g, size_image, hipMemcpyHostToDevice);
  hipMemcpy(gpu_blue, b, size_image, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_num_of_centroids),&num_of_centroids, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(gpu_size), &size, sizeof(int));
  	// Clearing centroids on gpu
	for(int i = 0; i < num_of_centroids; i++)
  {
			cpu_red_centroid[i] = 0;
			cpu_green_centroid[i] = 0;
			cpu_blue_centroid[i] = 0;
		}

		// Defining grid size
		int BLOCK_X, BLOCK_Y;
		BLOCK_X = ceil(width/BLOCK_SIZE);
		BLOCK_Y = ceil(height/BLOCK_SIZE);
		if(BLOCK_X > GRID_SIZE)
			BLOCK_X = GRID_SIZE;
		if(BLOCK_Y > GRID_SIZE)
			BLOCK_Y = GRID_SIZE;

	 	dim3 dimGRID(BLOCK_X,BLOCK_Y);
		dim3 dimBLOCK(BLOCK_SIZE,BLOCK_SIZE);

		printf("[3]launching K-Means Kernels..	\n");
		//Iteration of kmeans algorithm
		int num_iterations;
		for(int i = 0; i < num_of_iteration; i++)
  {
			num_iterations = i;
			cpu_flag=0;
			hipMemcpy(gpu_flag,&cpu_flag,sizeof(int),hipMemcpyHostToDevice);
			pop_gpu_arrays<<<1, dimBLOCK>>>(gpu_sum_red, gpu_sum_green, gpu_sum_blue,  gpu_points_in_cluster, gpu_update_red_centroid, gpu_update_green_centroid, gpu_update_blue_centroid);
			pop_gpu_cluster_array<<<dimGRID, dimBLOCK>>>(gpu_cluster_array);
			get_cluster_points<<< dimGRID, dimBLOCK >>> (gpu_red, gpu_green, gpu_blue,gpu_cluster_array);
			sum_pixels_in_cluster<<<dimGRID, dimBLOCK>>> (gpu_red, gpu_green, gpu_blue, gpu_sum_red, gpu_sum_green, gpu_sum_blue, gpu_cluster_array, gpu_points_in_cluster);
			update_centroids<<<1,dimBLOCK >>>(gpu_update_red_centroid, gpu_update_green_centroid, gpu_update_blue_centroid, gpu_sum_red, gpu_sum_green, gpu_sum_blue,  gpu_points_in_cluster,gpu_flag);
      hipMemcpy(cpu_red_centroid, gpu_update_red_centroid, size_for_centroids,hipMemcpyDeviceToHost);
      hipMemcpy(cpu_green_centroid, gpu_update_green_centroid, size_for_centroids,hipMemcpyDeviceToHost);
      hipMemcpy(cpu_blue_centroid, gpu_update_blue_centroid, size_for_centroids,hipMemcpyDeviceToHost);
      hipMemcpy(&cpu_flag, gpu_flag,sizeof(int),hipMemcpyDeviceToHost);
      hipMemcpyToSymbol(HIP_SYMBOL(gpu_red_centroid), cpu_red_centroid, size_for_centroids);
      hipMemcpyToSymbol(HIP_SYMBOL(gpu_green_centroid), cpu_green_centroid, size_for_centroids);
      hipMemcpyToSymbol(HIP_SYMBOL(gpu_blue_centroid), cpu_blue_centroid, size_for_centroids);
			if(cpu_flag==0)
				break;
		}

	 hipMemcpy(cpu_cluster_Array, gpu_cluster_array, size_image, hipMemcpyDeviceToHost);
	 hipMemcpy(cpu_points_in_cluster,  gpu_points_in_cluster, size_for_centroids, hipMemcpyDeviceToHost);

	printf("\tConverged in %d iterations.\n",num_iterations);
  // Estimating compression ratio
  display_compression_ratio(num_of_centroids);
  Mat uncompressed_image(input_image.rows,input_image.cols, CV_8UC3, Scalar(0, 0, 0));
   //8U means the 8-bit Usigned integer, C3 means 3 Channels for RGB color,
   //and Scalar(0, 0, 0) is the initial value for each pixel.
 	for (int i = 0; i < size; i++)
   {
     int x = i / input_image.cols;
     int y = i % input_image.cols;
     Vec3b intensity = uncompressed_image.at<Vec3b>(x,y);
     intensity.val[0]=cpu_blue_centroid[cpu_cluster_Array[i]];
     intensity.val[1]=cpu_green_centroid[cpu_cluster_Array[i]];
     intensity.val[2]=cpu_red_centroid[cpu_cluster_Array[i]];
     uncompressed_image.at<Vec3b>(x,y) = intensity;
   }
 	printf("[5]saving image...	\n");
  imwrite(remove_extension(argv[1]).append("cuda_uncompressed.jpg"),uncompressed_image);

	free(r);
	free(g);
	free(b);
	free(cpu_red_centroid);
	free(cpu_green_centroid);
	free(cpu_blue_centroid);
	free(cpu_cluster_Array);
	free(sum_red);
	free(sum_green);
	free(sum_blue);
	free(cpu_points_in_cluster);

	 hipFree(gpu_red);
	 hipFree(gpu_green);
	 hipFree(gpu_blue);
	 hipFree(gpu_update_red_centroid);
	 hipFree(gpu_update_green_centroid);
	 hipFree(gpu_update_blue_centroid);
	 hipFree(gpu_cluster_array);
	 hipFree(gpu_sum_red);
	 hipFree(gpu_sum_green);
	 hipFree(gpu_sum_blue);
	 hipFree( gpu_points_in_cluster);

	printf("[6]end...\n");
	return 0;
}
